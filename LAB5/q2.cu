#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1000              // Vector length (can be any size)
#define THREADS_PER_BLOCK 256

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int a[N], b[N], c[N];       // Host arrays
    int *d_a, *d_b, *d_c;       // Device arrays
    size_t size = N * sizeof(int);

    // Allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Calculate number of blocks needed
    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel
    vectorAdd<<<blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print first 10 results
    printf("Vector addition result (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}


