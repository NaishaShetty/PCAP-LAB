#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512 

__global__ void vectorAdd(int *a, int *b, int *c) {
    int i = threadIdx.x; // Each thread handles one element
    c[i] = a[i] + b[i];
}

int main() {
    int a[N], b[N], c[N];         // host arrays
    int *d_a, *d_b, *d_c;         // device arrays
    size_t size = N * sizeof(int);

    // Allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Copy input vectors to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel with 1 block and N threads
    vectorAdd<<<1, N>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print some of the results
    printf("Vector addition result (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

