#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main(void) {
    int a, b, c;                     // host copies of variables a, b & c
    int *d_a, *d_b, *d_c;            // device copies of variables a, b & c
    int size = sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Setup input values
    a = 3;
    b = 5;

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    printf("Result : %d\n", c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

